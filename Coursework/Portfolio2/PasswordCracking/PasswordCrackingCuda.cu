#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime_api.h>
#include <time.h>
/*
 Complie
   nvcc -o PasswordCrackingCuda PasswordCrackingCuda.cu
*/
__device__ int pass(char *password){

   char pass1[]="CY9829";
   char pass2[]="PR5782";
   char pass3[]="DE9811";
   char pass4[]="SH9819";

   char *p1 = password;
   char *p2 = password;
   char *p3 = password;
   char *p4 = password;

   char *pd1 = pass1;
   char *pd2 = pass2;
   char *pd3 = pass3;
   char *pd4 = pass4;

   while(*p1 == *pd1){
    
       if(*p1 == '\0'){
          return 1;
       }
       p1++;
       pd1++;
   }
   
    while(*p2 == *pd2){
    
       if(*p2 == '\0'){
          return 1;
       }
       p2++;
       pd2++;
   }

    while(*p3 == *pd3){
    
       if(*p3 == '\0'){
          return 1;
       }
       p3++;
       pd3++;
   }

    while(*p4 == *pd4){
    
       if(*p4 == '\0'){
          return 1;
       }
       p4++;
       pd4++;
   }
 return 0;
}

__global__ void kernel() {

 
char alphabet[26] = {'A','B','C','D','E','F','G','H','I','J','K','L','M','N','O','P','Q','R','S','T','U','V','W','X','Y','Z'};
  

char number[10] = {'0','1','2','3','4','5','6','7','8','9'};
  

  char password[7];
  password[6] = '\0';
  int c, d, e, f;

     for(c=0;c<10;c++){
      for(d=0; d<10; d++){
       for(e=0; e<10; e++){
        for(f=0; f<10; f++){

        password[0] = alphabet[blockIdx.x];
        password[1] = alphabet[threadIdx.x];
        password[2] = number[c];
        password[3] = number[d];
        password[4] = number[e];
        password[5] = number[f];

        if(pass(password)){
            printf("Password successfully cracked: %s\n", password);
        }

       }
      }
     }
    }
   }

/*
claculating the time difference.
*/
int time_difference(struct timespec *start, struct timespec *finish, long long int *difference)
 {
     long long int ds =  finish->tv_sec - start->tv_sec;
     long long int dn =  finish->tv_nsec - start->tv_nsec;

     if(dn < 0 )
     {
      ds--;
      dn += 1000000000;
      }

     *difference = ds * 1000000000 + dn;
    return !(*difference > 0);
}


/*
  Calulating the time
*/
int main(int argc, char *argv[])
{

    struct timespec start, finish;  
    long long int time_elapsed;

    clock_gettime(CLOCK_MONOTONIC, &start);

    kernel <<<26, 26>>>();

    hipDeviceSynchronize();


    clock_gettime(CLOCK_MONOTONIC, &finish);
    time_difference(&start, &finish, &time_elapsed);
     printf("Time elapsed was %lldns or %0.9lfs\n", time_elapsed,
                                           (time_elapsed/1.0e9));
  return 0;
}







